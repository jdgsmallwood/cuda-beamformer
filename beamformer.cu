#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MAGIC_STRING "\x93NUMPY"
#define MAGIC_STRING_LEN 6
#define MAX_LINE_LENGTH 1024
#define NUM_ANTENNAS 196
#define NUM_BEAMS 5
#define WARPS_PER_BLOCK 7
#define FULL_MASK 0xffffffff

typedef struct
{
    int index;
    float x_loc;
    float y_loc;
    float r;
} Antenna;

typedef struct
{
    float2 data[NUM_BEAMS * NUM_ANTENNAS];
} float2_beamarray;

typedef struct
{
    float2 data[NUM_BEAMS];
} float2_single_beamarray;

__global__ void beamform(const float2 *__restrict__ d_data, float2 *__restrict__ d_output, const float2_beamarray __restrict__ *d_weights_and_phase)
{
    __shared__ float2 partial_sum[NUM_BEAMS][WARPS_PER_BLOCK];
    float2 sum;
    const float2 data = d_data[blockIdx.x * blockDim.x + threadIdx.x];
    float2_single_beamarray weights_and_phase;

#pragma unroll
    for (int i = 0; i < NUM_BEAMS; i++)
    {
        // __ldg recommends that this be cached to read-only memory
        weights_and_phase.data[i] = __ldg(&(d_weights_and_phase->data[i * NUM_ANTENNAS + threadIdx.x]));
    }

#pragma unroll
    for (int beam = 0; beam < NUM_BEAMS; beam++)
    {
        sum.x = weights_and_phase.data[beam].y * data.x - data.y * weights_and_phase.data[beam].x;
        sum.y = data.x * weights_and_phase.data[beam].x + data.y * weights_and_phase.data[beam].y;

#pragma unroll
        for (int offset = 16; offset > 0; offset /= 2)
        {
            sum.x += __shfl_down_sync(FULL_MASK, sum.x, offset);
            sum.y += __shfl_down_sync(FULL_MASK, sum.y, offset);
        }

        // Is it the first thread in the warp? If so write to shared memory.
        if (threadIdx.x % 32 == 0)
        {
            partial_sum[beam][threadIdx.x / 32] = sum;
        }
    }

    __syncthreads();

    if (threadIdx.x < NUM_BEAMS)
    {
        sum = partial_sum[threadIdx.x][0];
#pragma unroll
        for (int i = 1; i < WARPS_PER_BLOCK; ++i)
        {
            sum.x += partial_sum[threadIdx.x][i].x;
            sum.y += partial_sum[threadIdx.x][i].y;
        }
        d_output[blockIdx.x * NUM_BEAMS + threadIdx.x] = sum;
    }
}

int extract_shape(const char *header, int *n_rows, int *n_cols)
{
    const char *shape_start = strstr(header, "'shape': (");
    if (!shape_start)
    {
        printf("Error: 'shape' not found in header.\n");
        return 0;
    }

    shape_start += strlen("'shape': (");

    int result = sscanf(shape_start, "%d, %d", n_rows, n_cols);
    if (result != 2)
    {
        printf("Error: Failed to parse shape.\n");
        return 0;
    }

    return 1;
}

void read_npy_file(const char *filename, float2 **data, int *n_rows, int *n_cols)
{
    FILE *file = fopen(filename, "rb");
    if (!file)
    {
        perror("Error opening file");
        return;
    }

    // Step 1: Check magic string
    unsigned char magic[MAGIC_STRING_LEN];
    fread(magic, 1, MAGIC_STRING_LEN, file);
    if (memcmp(magic, MAGIC_STRING, MAGIC_STRING_LEN) != 0)
    {
        printf("Not a valid .npy file.\n");
        fclose(file);
        return;
    }

    // Step 2: Read version numbers (2 bytes)
    unsigned char version[2];
    fread(version, 1, 2, file);
    printf("File version: %d.%d\n", version[0], version[1]);

    // Step 3: Read header length (2 bytes)
    unsigned char header_len_bytes[2];
    fread(header_len_bytes, 1, 2, file);
    short header_len = (header_len_bytes[0] | (header_len_bytes[1] << 8));

    printf("Header length: %d\n", header_len);

    // Step 4: Read header
    char *header = (char *)malloc(header_len + 1);
    if (!header)
    {
        perror("Memory allocation error");
        fclose(file);
        return;
    }
    fread(header, 1, header_len, file);
    header[header_len] = '\0'; // Null-terminate the string

    printf("Header: %s\n", header);

    if (!extract_shape(header, n_rows, n_cols))
    {
        printf("Error extracting shape.\n");
        free(header);
        fclose(file);
        return;
    }
    printf("Shape: %d x %d\n", *n_rows, *n_cols);

    // Step 5: Read data
    int data_size = (*n_rows) * (*n_cols);

    // Use pinned memory to improve performance.
    hipError_t err = hipHostMalloc((void **)data, data_size * sizeof(float2));
    if (err != hipSuccess)
    {
        perror("Memory allocation error");
        hipHostFree(header);
        fclose(file);
        return;
    }

    fread(*data, sizeof(float2), data_size, file);

    // Print first 5 elements of the first 5 antennae as a sanity check.
    printf("Data (first 5 complex numbers of each thread):\n");
    for (int i = 0; i < 5 && i < *n_rows; i++)
    {
        printf("Time Step %i:\n", i);
        for (int j = 0; j < 5 && j < *n_cols; j++)
        {
            printf("Complex %d: %.2f + %.2fi\n", j, (*data)[i * (*n_cols) + j].x, (*data)[i * (*n_cols) + j].y);
        }
    }

    free(header);
    fclose(file);
}

Antenna *read_antenna_map()
{
    FILE *file = fopen("/fred/oz002/jsmallwo/antenna_locations_only_used.csv", "r");
    if (!file)
    {
        perror("Could not open file");
        return NULL;
    }

    Antenna *antennae = NULL;
    hipError_t err = hipHostMalloc((void **)&antennae, NUM_ANTENNAS * sizeof(Antenna));

    char line[MAX_LINE_LENGTH];
    // skip header
    fgets(line, sizeof(line), file);
    int count = 0;
    // Read line by line
    while (fgets(line, sizeof(line), file))
    {
        char *token = strtok(line, ",");

        antennae[count].index = count;
        antennae[count].x_loc = atof(token);

        token = strtok(NULL, ",");
        antennae[count].y_loc = atof(token);
        token = strtok(NULL, ",");
        antennae[count].r = atof(token);
        count++;
    }

    fclose(file);
    return antennae;
}

int main()
{
    Antenna *antennas = read_antenna_map();

    float phase_offset[NUM_BEAMS * NUM_ANTENNAS];
    float weights[NUM_BEAMS * NUM_ANTENNAS];
    float2_beamarray weights_and_phase;

    float sin_phase, cos_phase;
    for (int beam = 0; beam < NUM_BEAMS; beam++)
    {
        for (int i = 0; i < NUM_ANTENNAS; i++)
        {
            // weights[i] = 1 / antennas[i].r;
            int offset = i * NUM_BEAMS + beam;
            int ant_offset = beam * NUM_ANTENNAS + i;
            weights[offset] = beam % 4; // Make things easy to start with.
            phase_offset[offset] = 0;

            sincosf(phase_offset[offset], &sin_phase, &cos_phase);
            weights_and_phase.data[ant_offset] = {sin_phase * weights[offset], cos_phase * weights[offset]};
        }
    }

    float2_beamarray *d_weights_and_phase = NULL;
    hipError_t err = hipMalloc((void **)&d_weights_and_phase, sizeof(float2_beamarray));
    if (err != hipSuccess)
    {
        printf("CUDA copy to phase offset symbol failed.\n");
        return -1;
    }

    const char *filename = "/fred/oz002/jsmallwo/antenna_data_transposed.npy";
    float2 *data = NULL;
    int n_rows, n_cols;
    read_npy_file(filename, &data, &n_rows, &n_cols);
    printf("data %f, %f\n", data[0].x, data[0].y);
    printf("data has shape %i x %i\n", n_rows, n_cols);

    float2 *d_data = NULL;
    err = hipMalloc((void **)&d_data, n_rows * n_cols * sizeof(float2));
    if (err != hipSuccess)
    {
        printf("CUDA memory allocation failed\n");
        hipHostFree(data);
        hipHostFree(antennas);
        hipFree(d_weights_and_phase);
        return -1;
    }

    float2 *d_output = NULL;
    err = hipMalloc((void **)&d_output, n_rows * NUM_BEAMS * sizeof(float2));
    if (err != hipSuccess)
    {
        printf("CUDA memory allocation failed\n");
        hipHostFree(data);
        hipHostFree(antennas);
        hipFree(d_data);
        hipFree(d_weights_and_phase);
        return -1;
    }

    float2 *output = NULL;
    err = hipHostMalloc((void **)&output, n_rows * NUM_BEAMS * sizeof(float2));
    if (err != hipSuccess)
    {
        printf("CUDA memory allocation failed\n");
        hipHostFree(data);
        hipHostFree(antennas);
        hipFree(d_data);
        hipFree(d_output);
        hipFree(d_weights_and_phase);
        return -1;
    }
    hipStream_t stream;
    err = hipStreamCreate(&stream);
    if (err != hipSuccess)
    {
        printf("CUDA stream creation failed\n");
        hipFree(d_data);
        hipFree(d_output);
        hipFree(d_weights_and_phase);
        hipHostFree(data);
        hipHostFree(output);
        hipHostFree(antennas);
        return -1;
    }

    err = hipMemcpyAsync(d_weights_and_phase, &weights_and_phase, sizeof(float2_beamarray), hipMemcpyHostToDevice, stream);

    if (err != hipSuccess)
    {
        printf("CUDA memory copy failed\n");
        hipFree(d_data);
        hipFree(d_weights_and_phase);
        hipFree(d_output);
        hipHostFree(data);
        hipHostFree(output);
        hipHostFree(antennas);
        hipStreamDestroy(stream);
        return -1;
    }
    err = hipMemcpyAsync(d_data, data, n_rows * n_cols * sizeof(float2), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        printf("CUDA memory copy failed\n");
        hipFree(d_data);
        hipFree(d_weights_and_phase);
        hipFree(d_output);
        hipHostFree(data);
        hipHostFree(output);
        hipHostFree(antennas);
        hipStreamDestroy(stream);
        return -1;
    }
    // Check for kernel execution errors
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
    }

    err = hipStreamSynchronize(stream);
    if (err != hipSuccess)
    {
        printf("CUDA stream synchronization failed\n");
    }

    beamform<<<dim3(n_rows, 1), NUM_ANTENNAS, 0, stream>>>(d_data, d_output, d_weights_and_phase);

    err = hipStreamSynchronize(stream);
    if (err != hipSuccess)
    {
        printf("CUDA stream synchronization failed\n%s\n", hipGetErrorString(err));
    }

    err = hipMemcpyAsync(output, d_output, n_rows * NUM_BEAMS * sizeof(float2), hipMemcpyDeviceToHost, stream);
    if (err != hipSuccess)
    {
        printf("CUDA memory copy failed\n");
        hipFree(d_weights_and_phase);
        hipFree(d_data);
        hipFree(d_output);
        hipHostFree(data);
        hipHostFree(output);
        hipStreamDestroy(stream);
        return -1;
    }

    err = hipStreamSynchronize(stream);
    if (err != hipSuccess)
    {
        printf("CUDA stream synchronization failed\n%s\n", hipGetErrorString(err));
    }

    for (int beam = 0; beam < NUM_BEAMS; beam++)
    {
        printf("First 5 values of beam %i are...\n", beam);
        for (int i = 0; i < 5; i++)
        {
            printf("%f + %fi\n", output[i * n_rows + beam].x, output[i * n_rows + beam].y);
        }

    }

    hipFree(d_data);
    hipFree(d_weights_and_phase);
    hipFree(d_output);
    hipFree(d_output);
    hipHostFree(antennas);
    hipHostFree(data);
    hipHostFree(output);
    hipHostFree(output);
    hipStreamDestroy(stream);
    return 0;
}