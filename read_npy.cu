#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAGIC_STRING "\x93NUMPY"
#define MAGIC_STRING_LEN 6
#define MAX_LINE_LENGTH 1024
#define NUM_ANTENNAS 196
#define NUM_BEAMS 5
#define WARPS_PER_BLOCK 7
#define FULL_MASK 0xffffffff



typedef struct
{
    int index;
    float x_loc;
    float y_loc;
    float r;
} Antenna;

__constant__ float d_weights[NUM_ANTENNAS * NUM_BEAMS];
__constant__ float d_phase_offset[NUM_ANTENNAS * NUM_BEAMS];


__global__ void beamform(float2 *d_data, int n_rows, int n_cols, float2 *d_output)
{
    __shared__ float2 shared_sum[WARPS_PER_BLOCK];
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Each data point will have NUM_ANTENNAS threads associated with it
    // So we can figure out which time step and antenna we are associated with.

    float2 sum;
    sum.x = 0;
    sum.y = 0;

    if (idx < n_cols * n_rows)
    {
        // printf("Antennae %i: weight %f phase_offset %f\n", idx, weights[idx], phase_offset[idx]);
        const float2 data = d_data[idx];
        const int offset_to_read = blockIdx.y * NUM_ANTENNAS + threadIdx.x;
        const float weight = d_weights[offset_to_read];
        const float phase = d_phase_offset[offset_to_read];
        sum.x += weight * phase * data.x;
        sum.y +=weight * phase * data.y;
    }

    for (int offset = 16; offset > 0; offset /= 2)
    {
        sum.x += __shfl_down_sync(FULL_MASK, sum.x, offset);
        sum.y += __shfl_down_sync(FULL_MASK, sum.y, offset);
    }

    // Is it the first thread in the warp?
    if (threadIdx.x % 32== 0)
    {
        // this is the warp number. 
        shared_sum[(int)(threadIdx.x / 32)] = sum;
    }

    __syncthreads();
    sum.x = 0;
    sum.y = 0;
    if (threadIdx.x < WARPS_PER_BLOCK)
    {
        sum = shared_sum[threadIdx.x];
        for (int offset = 16; offset > 0; offset /= 2)
        {
            // can improve this by 1 loop.
            sum.x += __shfl_down_sync(FULL_MASK, sum.x, offset);
            sum.y += __shfl_down_sync(FULL_MASK, sum.y, offset);
        }
    }

    if (threadIdx.x == 0)
    {
        d_output[blockIdx.y * n_rows + blockIdx.x] = sum;
    }
}

int extract_shape(const char *header, int *n_rows, int *n_cols)
{
    const char *shape_start = strstr(header, "'shape': (");
    if (!shape_start)
    {
        printf("Error: 'shape' not found in header.\n");
        return 0;
    }

    shape_start += strlen("'shape': (");

    int result = sscanf(shape_start, "%d, %d", n_rows, n_cols);
    if (result != 2)
    {
        printf("Error: Failed to parse shape.\n");
        return 0;
    }

    return 1;
}

void read_npy_file(const char *filename, float2 **data, int *n_rows, int *n_cols)
{
    FILE *file = fopen(filename, "rb");
    if (!file)
    {
        perror("Error opening file");
        return;
    }

    // Step 1: Check magic string
    unsigned char magic[MAGIC_STRING_LEN];
    fread(magic, 1, MAGIC_STRING_LEN, file);
    if (memcmp(magic, MAGIC_STRING, MAGIC_STRING_LEN) != 0)
    {
        printf("Not a valid .npy file.\n");
        fclose(file);
        return;
    }

    // Step 2: Read version numbers (2 bytes)
    unsigned char version[2];
    fread(version, 1, 2, file);
    printf("File version: %d.%d\n", version[0], version[1]);

    // Step 3: Read header length (2 bytes)
    unsigned char header_len_bytes[2];
    fread(header_len_bytes, 1, 2, file);
    short header_len = (header_len_bytes[0] | (header_len_bytes[1] << 8));

    printf("Header length: %d\n", header_len);

    // Step 4: Read header
    char *header = (char *)malloc(header_len + 1);
    if (!header)
    {
        perror("Memory allocation error");
        fclose(file);
        return;
    }
    fread(header, 1, header_len, file);
    header[header_len] = '\0'; // Null-terminate the string

    printf("Header: %s\n", header);

    if (!extract_shape(header, n_rows, n_cols))
    {
        printf("Error extracting shape.\n");
        free(header);
        fclose(file);
        return;
    }
    printf("Shape: %d x %d\n", *n_rows, *n_cols);

    // Step 5: Read data
    int data_size = (*n_rows) * (*n_cols);

    // Use pinned memory to improve performance.
    hipError_t err = hipHostMalloc((void **)data, data_size * sizeof(float2), hipHostMallocDefault);
    if (err != hipSuccess)
    {
        perror("Memory allocation error");
        hipHostFree(header);
        fclose(file);
        return;
    }

    fread(*data, sizeof(float2), data_size, file);

    // Print first 5 elements of the first 5 antennae as a sanity check.
    printf("Data (first 5 complex numbers of each thread):\n");
    for (int i = 0; i < 5 && i < *n_rows; i++)
    {
        printf("Time Step %i:\n", i);
        for (int j = 0; j < 5 && j < *n_cols; j++)
        {
            printf("Complex %d: %.2f + %.2fi\n", j, (*data)[i * (*n_cols) + j].x, (*data)[i * (*n_cols) + j].y);
        }
    }

    free(header);
    fclose(file);
}

Antenna *read_antenna_map()
{
    FILE *file = fopen("../antenna_locations_only_used.csv", "r");
    if (!file)
    {
        perror("Could not open file");
        return NULL;
    }

    Antenna *antennae = NULL;
    hipError_t err = hipHostMalloc((void **)&antennae, NUM_ANTENNAS * sizeof(Antenna), hipHostMallocDefault);

    char line[MAX_LINE_LENGTH];
    // skip header
    fgets(line, sizeof(line), file);
    int count = 0;
    // Read line by line
    while (fgets(line, sizeof(line), file))
    {
        char *token = strtok(line, ",");

        antennae[count].index = count;
        antennae[count].x_loc = atof(token);

        token = strtok(NULL, ",");
        antennae[count].y_loc = atof(token);
        token = strtok(NULL, ",");
        antennae[count].r = atof(token);
        count++;
    }

    fclose(file);
    return antennae;
}

int main()
{
    Antenna *antennas = read_antenna_map();

    float phase_offset[NUM_BEAMS * NUM_ANTENNAS];
    float weights[NUM_BEAMS * NUM_ANTENNAS];
    for (int beam = 0; beam < NUM_BEAMS; beam++) {
    for (int i = 0; i < NUM_ANTENNAS; i++)
    {
        // weights[i] = 1 / antennas[i].r;
        weights[beam*NUM_ANTENNAS + i] = beam; // Make things easy to start with.
        phase_offset[beam * NUM_ANTENNAS + i] = 1;
    }
}
    
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(d_weights), weights, sizeof(float) * NUM_ANTENNAS * NUM_BEAMS);
    if (err != hipSuccess)
    {
        printf("CUDA copy to weights symbol failed.\n");
        return -1;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(d_phase_offset), phase_offset, sizeof(float) * NUM_ANTENNAS * NUM_BEAMS);
    if (err != hipSuccess)
    {
        printf("CUDA copy to phase offset symbol failed.\n");
        return -1;
    }

    const char *filename = "../antenna_data_transposed.npy";
    float2 *data = NULL;
    int n_rows, n_cols;
    read_npy_file(filename, &data, &n_rows, &n_cols);
    printf("data %f, %f\n", data[0].x, data[0].y);
    printf("data has shape %i x %i\n", n_rows, n_cols);

    float2 *d_data = NULL;
    err = hipMalloc((void **)&d_data, n_rows * n_cols * sizeof(float2));
    if (err != hipSuccess)
    {
        printf("CUDA memory allocation failed\n");
        hipHostFree(data);
        hipHostFree(antennas);
        hipFree(d_weights);
        hipFree(d_phase_offset);
        return -1;
    }

    float2 *d_output = NULL;
    err = hipMalloc((void **)&d_output, n_rows * NUM_BEAMS * sizeof(float2));
    if (err != hipSuccess)
    {
        printf("CUDA memory allocation failed\n");
        hipHostFree(data);
        hipHostFree(antennas);
        hipFree(d_data);
        hipFree(d_weights);
        hipFree(d_phase_offset);
        return -1;
    }

    float2 *output = NULL;
    err = hipHostMalloc((void **)&output, n_rows * NUM_BEAMS * sizeof(float2), hipHostMallocDefault);
    if (err != hipSuccess)
    {
        printf("CUDA memory allocation failed\n");
        hipHostFree(data);
        hipHostFree(antennas);
        hipFree(d_data);
        hipFree(d_output);
        hipFree(d_phase_offset);
        hipFree(d_weights);
        return -1;
    }

    hipStream_t stream;
    err = hipStreamCreate(&stream);
    if (err != hipSuccess)
    {
        printf("CUDA stream creation failed\n");
        hipFree(d_data);
        hipFree(d_output);
        hipFree(d_weights);
        hipFree(d_phase_offset);
        hipHostFree(data);
        hipHostFree(output);
        hipHostFree(antennas);
        return -1;
    }

    err = hipMemcpyAsync(d_data, data, n_rows * n_cols * sizeof(float2), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        printf("CUDA memory copy failed\n");
        hipFree(d_data);
        hipFree(d_weights);
        hipFree(d_phase_offset);
        hipFree(d_output);
        hipHostFree(data);
        hipHostFree(output);
        hipHostFree(antennas);
        hipStreamDestroy(stream);
        return -1;
    }
    // Check for kernel execution errors
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
    }

    err = hipStreamSynchronize(stream);
    if (err != hipSuccess)
    {
        printf("CUDA stream synchronization failed\n");
    }

    beamform<<<dim3(n_rows, NUM_BEAMS), NUM_ANTENNAS, 0, stream>>>(d_data, n_rows, n_cols, d_output);

    err = hipStreamSynchronize(stream);
    if (err != hipSuccess)
    {
        printf("CUDA stream synchronization failed\n%s\n", hipGetErrorString(err));
    }

    err = hipMemcpyAsync(output, d_output, n_rows * NUM_BEAMS * sizeof(float2), hipMemcpyDeviceToHost, stream);
    if (err != hipSuccess)
    {
        printf("CUDA memory copy failed\n");
        hipFree(d_weights);
        hipFree(d_data);
        hipFree(d_phase_offset);
        hipFree(d_output);
        hipHostFree(data);
        hipHostFree(output);
        hipStreamDestroy(stream);
        return -1;
    }

    err = hipStreamSynchronize(stream);
    if (err != hipSuccess)
    {
        printf("CUDA stream synchronization failed\n%s\n", hipGetErrorString(err));
    }

    for (int beam = 0; beam < NUM_BEAMS; beam++) {
    printf("First 5 values of beam %i are...\n", beam);
    for (int i = 0; i < 5; i++)
    {
        printf("%f + %fi\n", output[beam * n_rows + i].x, output[beam * n_rows + i].y);
    }

    printf("Last value is...\n");
    //printf("%f + %fi\n", output[beam * (n_rows + 1) - 1].real, output[beam * (n_rows + 1) - 1].imag);
}

    hipFree(d_data);
    hipFree(d_weights);
    hipFree(d_phase_offset);
    hipFree(d_output);
    hipHostFree(antennas);
    hipHostFree(data);
    hipHostFree(output);
    hipStreamDestroy(stream);
    return 0;
}